#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "helper_structs.h"
#include "hip/hip_vector_types.h"

__global__ void copy_particle_positions(float* translations, Particle* particles, int N) {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < N) {
		translations[tid*3 + 0] = particles[tid].pos.x;
        translations[tid*3 + 1] = particles[tid].pos.y;
        translations[tid*3 + 2] = particles[tid].pos.z;
	}
}