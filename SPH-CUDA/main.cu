#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "helper_structs.h"
#include "file_manager.h"
#include "error_checker.h"

#include "cell_structure.cuh"
#include "calculate_density.cuh"
#include "calculate_force.cuh"
#include "integrators.cuh"

#include "visualizer.h"
#include "particlePositionCopy.cu"
#include <cuda_gl_interop.h>

void initializeParticles(std::vector<Particle>&, Parameters&);

int main() {
	FileManager file_manager("parameter_files", "params_0.par");
	Parameters params = file_manager.readParams();

	std::vector<Particle> particles;
	initializeParticles(particles, params);

	std::vector<int> cell_list(params.cell_num, -1);
	std::vector<int> particle_list(params.particle_num, -1);

	/* Allocate memory on device */
	Particle* d_particles;
	float3* d_force_buffer;
	float* d_density_buffer;
	int* d_particle_list, * d_cell_list;
	size_t bytes_vec = sizeof(float) * params.particle_num;
	size_t bytes_vec3 = sizeof(float3) * params.particle_num;
	size_t bytes_struct = sizeof(Particle) * params.particle_num;
	size_t bytes_particle_list = sizeof(int) * params.particle_num;
	size_t bytes_cell_list = sizeof(int) * params.cell_num;
	checkError(hipMalloc((void**)&d_particle_list, bytes_particle_list));
	checkError(hipMalloc((void**)&d_cell_list, bytes_cell_list));
	checkError(hipMalloc((void**)&d_particles, bytes_struct));
	checkError(hipMalloc(&d_force_buffer, bytes_vec3));
	checkError(hipMalloc(&d_density_buffer, bytes_vec));

	/* Copy data to device */
	checkError(hipMemcpy(d_particles, particles.data(), bytes_struct, hipMemcpyHostToDevice));
	checkError(hipMemcpy(d_particle_list, particle_list.data(), bytes_particle_list, hipMemcpyHostToDevice));
	checkError(hipMemcpy(d_cell_list, cell_list.data(), bytes_cell_list, hipMemcpyHostToDevice));

	/* Visualization init */
	Visualizer vis(params.particle_num, params.particle_radius, params.min_box_bound.x, params.min_box_bound.y, params.min_box_bound.z,
		params.max_box_bound.x, params.max_box_bound.y, params.max_box_bound.z);

	struct hipGraphicsResource* positionsVBO_CUDA = NULL;
	checkError(hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, vis.vertexArray, cudaGraphicsMapFlagsWriteDiscard));

	std::cout << "Simulation started" << std::endl;
	while (!glfwWindowShouldClose(vis.window)) {

		// Start time measurement
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		/* Set all entries of cell list to -1 */
		reset_cell_list << <params.thread_groups_cell, params.threads_per_group >> > (d_cell_list, params.cell_num);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		/* Initialize cell list and particle list */
		assign_to_cells << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list,
			params.particle_num, params.cell_dims, params.min_box_bound, params.h_inv);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		/* Calculate densities */
		calculate_density << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list, d_density_buffer,
			params.cell_dims, params.min_box_bound, params.particle_num, params.h, params.h2, params.h_inv, params.const_poly6, params.mass, params.p0);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		/* Calculate forces */
		calculate_force << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list, d_force_buffer, d_density_buffer, params.cell_dims, params.min_box_bound,
			params.particle_num, params.h, params.h_inv, params.const_spiky, params.const_visc, params.const_surf, params.mass, params.k, params.e, params.p0, params.s, params.g);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		/* Integrate new positions and velocities */
		integrate_symplectic_euler << <params.thread_groups_part, params.threads_per_group >> >
			(d_particles, d_force_buffer, params.time_step, params.particle_num, params.min_box_bound, params.max_box_bound, params.damping);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		// Stop time measurement
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "ms" << std::endl;

		/* Visualization update */
		float* vertexPointer;
		// Map the buffer to CUDA
		checkError(hipGraphicsMapResources(1, &positionsVBO_CUDA));
		size_t numBytes;
		checkError(hipGraphicsResourceGetMappedPointer((void **)&vertexPointer, &numBytes, positionsVBO_CUDA));
		// Run kernel
		copy_particle_positions<<<params.thread_groups_part, params.threads_per_group>>>((float*)vertexPointer, d_particles, params.particle_num);
		// Unmap the buffer
		checkError(hipGraphicsUnmapResources(1, &positionsVBO_CUDA));

		vis.draw(params.particle_num);

		// Stop time measurement
		end = std::chrono::steady_clock::now();
		std::cout << "Time2 = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "ms" << std::endl;

	}

	std::cout << "Simulation finished" << std::endl;

	/* Free memory on device */
	checkError(hipFree(d_particles));
	checkError(hipFree(d_force_buffer));
	checkError(hipFree(d_particle_list));
	checkError(hipFree(d_cell_list));

	/* Visualization end */
	vis.end();
}


/* Spawns particles in a cubic shape */
void initializeParticles(std::vector<Particle>& particles, Parameters& p) {
	// Calculate shift in order to spawn the cubic shape in the center of the box
	// Shift equals half of the length of the cubic shape
	float shift = (p.edge_length * p.spawn_dist) / 2;

	for (int i = 0; i < p.particle_num; i++) {
		
		// Calculate cubic shape
		float x = (i % p.edge_length) * p.spawn_dist;
		float y = ((i / p.edge_length) % p.edge_length) * p.spawn_dist;
		float z = (i / (p.edge_length * p.edge_length)) * p.spawn_dist;

		// Add offsets
		x += p.spawn_offset.x - shift;
		y += p.spawn_offset.y - shift;
		z += p.spawn_offset.z - shift;

		particles.emplace_back(make_float3(x, y, z), make_float3(0., 0., 0.));
	}
}